#include "hip/hip_runtime.h"
#include "aritmetica.h"
__global__
void FourKernel(double* y_even, double* y_odd, double* y, int tam) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i<tam / 2) {
        y[i] = (y_even[i] + y_odd[i]);
        int indice = i + tam / 2;
        y[indice] = (y_even[i] - y_odd[i]);
    }
}
void fourpara(double* y, double* y_even, double* y_odd, double* X, int tam)
{
    int size = tam* sizeof(double);
    double *d_y, *d_y_even, *d_y_odd;
    hipMalloc((void **)&d_y_even, size / 2);
    hipMemcpy(d_y_even, y_even, size / 2, hipMemcpyHostToDevice);
    hipMalloc((void **)&d_y_odd, size / 2);
    hipMemcpy(d_y_odd, y_odd, size / 2, hipMemcpyHostToDevice);
    hipMalloc((void **)&d_y, size);
    FourKernel << < ceil(tam / 256.0), 256 >> > (d_y_even, d_y_odd, d_y, tam);
    hipMemcpy(y, d_y, size, hipMemcpyDeviceToHost);
    hipFree(d_y);
    hipFree(d_y_even);
    hipFree(d_y_odd);
}
double* aritmetica::four(double* data, double n, int tam)
{

    double * even = (double *)malloc((tam / 2)*sizeof(double));
    double * odd = (double *)malloc((tam / 2)*sizeof(double));
    double * y_even = (double *)malloc((tam / 2)*sizeof(double));
    double * y_odd = (double *)malloc((tam / 2)*sizeof(double));
    if (tam == 1) return data;
    for (int i = 0; i < tam / 2; i++)
    {
        even[i] = data[i * 2];
        odd[i] = data[i * 2 + 1];
    }
    y_even = four(even, pow(n, 2), tam / 2);
    y_odd = four(odd, pow(n, 2), tam / 2);
    double X = 1;
    double* y = new double[tam];
    fourpara(y, y_even, y_odd, &X, tam);
    return y;
}

ZZ Blum(long n)
{
    ZZ N,semilla, p, q, bits,x, res, temp;
    aritmetica op;

    p = 346176527;
    q = 7171153257;

    N = p * q;
    clock_t t;
    t=clock();
    semilla =t;
    x = semilla%N;
    res = 0, bits = 0;

    #pragma omp parallel for
    for(int i=n; i>0; i--)
    {
        x = op.pow(x, to_ZZ(2), N);
        bits = x-((x>>1)<<1);
        power(temp,to_ZZ(2),(i-1));
        res += bits*temp;
    }
    return res;
}
ZZ aleatorioBits(long long i)
{
    ZZ d =Blum(i);
    // cout<<d<<endl;
    if((d &1)==0)
        return d+1;
    else
        return d;
}
ZZ generaPrimo(long long bits)
{
    ZZ n =aleatorioBits(bits);
    while(MillerWitness((n),to_ZZ(80))==0)
        n=aleatorioBits(bits);
    return n;
}





