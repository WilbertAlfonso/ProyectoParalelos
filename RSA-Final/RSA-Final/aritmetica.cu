#include "hip/hip_runtime.h"
#include "aritmetica.h"
__global__
void FourKernel(double* y_even, double* y_odd, double* y, int tam){
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i<tam / 2){
		y[i] = (y_even[i] + y_odd[i]);
		int indice = i + tam / 2;
		y[indice] = (y_even[i] - y_odd[i]);
	}
}
void fourpara(double* y, double* y_even, double* y_odd, double* X, int tam)
{
	int size = tam* sizeof(double);
	double *d_y, *d_y_even, *d_y_odd;
	hipMalloc((void **)&d_y_even, size / 2);
	hipMemcpy(d_y_even, y_even, size / 2, hipMemcpyHostToDevice);
	hipMalloc((void **)&d_y_odd, size / 2);
	hipMemcpy(d_y_odd, y_odd, size / 2, hipMemcpyHostToDevice);
	hipMalloc((void **)&d_y, size);
	FourKernel << < ceil(tam / 256.0), 256 >> > (d_y_even, d_y_odd, d_y, tam);
	hipMemcpy(y, d_y, size, hipMemcpyDeviceToHost);
	hipFree(d_y); hipFree(d_y_even); hipFree(d_y_odd);
}
double* aritmetica::four(double* data, double n, int tam)
{

	double * even = (double *)malloc((tam / 2)*sizeof(double));
	double * odd = (double *)malloc((tam / 2)*sizeof(double));
	double * y_even = (double *)malloc((tam / 2)*sizeof(double));
	double * y_odd = (double *)malloc((tam / 2)*sizeof(double));
	if (tam == 1) return data;
	for (int i = 0; i < tam / 2; i++)
	{
		even[i] = data[i * 2];
		odd[i] = data[i * 2 + 1];
	}
	y_even = four(even, pow(n, 2), tam / 2);
	y_odd = four(odd, pow(n, 2), tam / 2);
	double X = 1;
	double* y = new double[tam];
	fourpara(y, y_even, y_odd, &X, tam);
	return y;
}
