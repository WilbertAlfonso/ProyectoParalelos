#include "hip/hip_runtime.h"
#include "aritmetica.h"
__global__
void FFTKernel(double* y_even, double* y_odd, double* y, double omega, int tam){
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i<tam / 2){
		int n = pow((double)omega, (double)i);
		y[i] = (y_even[i] + n*y_odd[i]);
		int indice = i + tam / 2;
		y[indice] = (y_even[i] - n*y_odd[i]);
	}
}
void fourpara(double* y, double* y_even, double* y_odd, double* X, int tam)
{
	int size = tam* sizeof(double);
	double *d_y, *d_y_even, *d_y_odd;
	hipMalloc((void **)&d_y_even, size / 2);
	hipMemcpy(d_y_even, y_even, size / 2, hipMemcpyHostToDevice);
	hipMalloc((void **)&d_y_odd, size / 2);
	hipMemcpy(d_y_odd, y_odd, size / 2, hipMemcpyHostToDevice);
	hipMalloc((void **)&d_y, size);
	FFTKernel << < ceil(tam / 256.0), 256 >> > (d_y_even, d_y_odd, d_y, *X, tam);
	hipMemcpy(y, d_y, size, hipMemcpyDeviceToHost);
	hipFree(d_y); hipFree(d_y_even); hipFree(d_y_odd);
}
double* aritmetica::four(double* data, double n, int tam)
{

	double * even = (double *)malloc((tam / 2)*sizeof(double));
	double * odd = (double *)malloc((tam / 2)*sizeof(double));
	double * y_even = (double *)malloc((tam / 2)*sizeof(double));
	double * y_odd = (double *)malloc((tam / 2)*sizeof(double));
	if (tam == 1) return data;
	for (int i = 0; i < tam / 2; i++)
	{
		even[i] = data[i * 2];
		odd[i] = data[i * 2 + 1];
	}
	y_even = four(even, n, tam / 2);
	y_odd = four(odd, n, tam / 2);
	double* y = new double[tam];
	fourpara(y, y_even, y_odd, &n, tam);
	return y;
}

__global__
void FFTIKernel(double* y_even, double* y_odd, double* y, double omega, int tam){
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i<tam / 2){
		int n = pow((double)omega, (double)i);
		y[i] = (y_even[i] + n*y_odd[i]) / 2;
		int indice = i + tam / 2;
		y[indice] = (y_even[i] - n*y_odd[i]) / 2;
	}
}
void fourIpara(double* y, double* y_even, double* y_odd, double* X, int tam)
{
	int size = tam* sizeof(double);
	double *d_y, *d_y_even, *d_y_odd;
	hipMalloc((void **)&d_y_even, size / 2);
	hipMemcpy(d_y_even, y_even, size / 2, hipMemcpyHostToDevice);
	hipMalloc((void **)&d_y_odd, size / 2);
	hipMemcpy(d_y_odd, y_odd, size / 2, hipMemcpyHostToDevice);
	hipMalloc((void **)&d_y, size);
	FFTIKernel << < ceil(tam / 256.0), 256 >> > (d_y_even, d_y_odd, d_y, *X, tam);
	hipMemcpy(y, d_y, size, hipMemcpyDeviceToHost);
	hipFree(d_y); hipFree(d_y_even); hipFree(d_y_odd);
}
double* aritmetica::fourI(double* data, double n, int tam)
{

	double * even = (double *)malloc((tam / 2)*sizeof(double));
	double * odd = (double *)malloc((tam / 2)*sizeof(double));
	double * y_even = (double *)malloc((tam / 2)*sizeof(double));
	double * y_odd = (double *)malloc((tam / 2)*sizeof(double));
	if (tam == 1) return data;
	for (int i = 0; i < tam / 2; i++)
	{
		even[i] = data[i * 2];
		odd[i] = data[i * 2 + 1];
	}
	y_even = fourI(even, n, tam / 2);
	y_odd = fourI(odd, n, tam / 2);
	double* y = new double[tam];
	fourIpara(y, y_even, y_odd, &n, tam);
	return y;
}

double* aritmetica::Mult(double* X, double* Y, int numbits)
{
	double* MX = (double*)malloc(numbits*sizeof(double));
	double* MY = (double*)malloc(numbits*sizeof(double));
	memset(MX, 0, numbits*sizeof(double));
	memset(MY, 0, numbits*sizeof(double));
	memcpy(MX, X, 2 * sizeof(*X));
	memcpy(MY, Y, 2 * sizeof(*X));
	for (int i = 0; i < numbits; i++)
	{
		cout << MY[i] << ";";
	}
	cout << endl;
	cout << endl;
	double* FX = four(MX, -1, numbits);

	double* FY = four(MY, -1, numbits);
	for (int i = 0; i < numbits; i++)
	{
		cout << FY[i] << ";";
	}
	cout << endl;
	cout << endl;
	double* FYI = fourI(FY, -1, numbits);
	for (int i = 0; i < numbits; i++)
	{
		cout << FYI[i] << ";";
	}
	cout << endl;
	cout << endl;
	double* Resp = new double[numbits];
	for (int i = 0; i < numbits; i++)
	{
		Resp[i] = FX[i] * FY[i];
		cout << Resp[i] << ";";
	}
	cout << endl;
	cout << endl;
	double* Inv = fourI(Resp, -1, numbits);
	for (int i = 0; i < numbits; i++)
	{
		cout << Inv[i] << ";";
	}
	return X;
}
ZZ aritmetica::powM(ZZ a, ZZ m, ZZ modulo)
{

	ZZ respuesta;
	respuesta = 1;
	ZZ x;
	x = a;
	while (m != 0)
	{

		if ((m & 1) == 1)
		{
			respuesta = (respuesta*x) % modulo;
			//cout<<"respuesta_ "<<respuesta<<" x: "<<x<<endl;


		}
		x = (x*x) % modulo;
		m >>= 1;
		// cout<<"m: "<<m<<" x: "<<x<<" respuesta: "<<respuesta<<endl;
		//if(mod(x,modulo)==1) break;

	}
	//cout<<endl;
	return respuesta;
}
ZZ aritmetica::Blum(long n)
{
    ZZ N,semilla, p, q, bits,x, res, temp;

    p = 7171153257;
    q =5;

    N = p * q;
    clock_t t;
    t=clock();
    semilla =t;
    x = semilla%N;
    res = 0, bits = 0;

    #pragma omp parallel for
    for(int i=n; i>0; i--)
    {
        x = powM(x, to_ZZ(2), N);
        bits = x-((x>>1)<<1);
        power(temp,to_ZZ(2),(i-1));
        res += bits*temp;
    }
    return res;
}
ZZ aritmetica::aleatorioBits(long long i)
{
    ZZ d =Blum(i);
    // cout<<d<<endl;
    if((d &1)==0)
        return d+1;
    else
        return d;
}
/*ZZ aritmetica::generaPrimo(long long bits)
{
    ZZ n =aleatorioBits(bits);
    while(MillerWitness((n),to_ZZ(80))==0)
        n=aleatorioBits(bits);
    return n;
}*/
ZZ aritmetica::generaPrimo(long long bits)
{
    ZZ n;
   
   do {
        n=aleatorioBits(bits);
        //cout<<n<<endl<<endl;
        //cout<<ProbPrime(n)<<endl<<endl;
    } while(ProbPrime(n)==0);
    return n;
}





